#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "bitonic.h"

__device__ void swap(int *array, int idx, int partner)
{
    int temp;
    temp = array[idx];
    array[idx] = array[partner];
    array[partner] = temp;
}

__global__ void exchangeKernel(int *array, int size, int group_size, int distance)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int idx = (tid / distance) * distance * 2 + (tid % distance);
    int partner = idx ^ distance;
    bool sort_descending = idx & group_size;

    if (idx < size && partner < size && idx < partner)
    { // ensure bounds are checked before accessing array

        if (!sort_descending && array[idx] > array[partner])
        {
            // keep min elements
            swap(array, idx, partner);
        }
        if (sort_descending && array[idx] < array[partner])
        {
            // keep max elements
            swap(array, idx, partner);
        }
    }
}

void bitonicSort(int *array, int size)
{
    // GPU PARAMETERS
    int threads_per_block = 1024;                   // max threads
    int blocks_per_grid = size / threads_per_block; // more if its not divided evenly

    for (int group_size = 2; group_size <= size; group_size <<= 1)
    { // group_size doubles in each reccursion

        for (int distance = group_size >> 1; distance > 0; distance >>= 1)
        { // half distance

            exchangeKernel<<<blocks_per_grid, threads_per_block>>>(array, size, group_size, distance);
            // debbuging
            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("CUDA Error: %s\n", hipGetErrorString(err));
            }
            hipDeviceSynchronize();
        }
    }
}

void print(int *array, int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%2d ", array[i]);
    }
    printf("\n");
}

void evaluateResult(int *array, int size)
{

    bool is_Sorted = true;
    for (int i = 0; i < size - 1; i++)
    {
        if (array[i] > array[i + 1])
        {
            is_Sorted = false;
            break;
        }
    }

    if (is_Sorted)
    {
        printf("Sorted array\n");
    }
    else
        printf("Array is not sorted!!\n");
}