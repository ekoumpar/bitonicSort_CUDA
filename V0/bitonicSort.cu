#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "bitonic.h"

__global__ void exchangeKernel(int *array, int size, int group_size, int distance)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int partner = idx ^ distance;
    bool sort_descending = idx & group_size;
	
    if (idx >= size || partner >= size) return;
    if ((!sort_descending && idx < partner) || (sort_descending && idx > partner))
    {
        // keep min elements
        if (array[idx] > array[partner])
        {
          int temp;
	        temp = array[idx];
    	    array[idx] = array[partner];
    	    array[partner] = temp;
	}    
    }
    else
    { // keep max elements
           
	 if (array[idx] < array[partner])
         {
            int temp;
            temp = array[idx];
            array[idx] = array[partner];
            array[partner] = temp;

        }
    }
}

void bitonicSort(int *array, int size)
{
    // GPU PARAMETERS
    int threads_per_block = 1024; // max threads
    int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;   //more if its not divided evenly


    for (int group_size = 2; group_size <= size; group_size <<= 1)
    { // group_size doubles in each reccursion

        for (int distance = group_size >> 1; distance > 0; distance >>= 1)
        { // half distance

          exchangeKernel<<<blocks_per_grid, threads_per_block>>>(array, size, group_size, distance);
	        //debbuging
          hipError_t err = hipGetLastError();
          if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
          }
	        hipDeviceSynchronize();
        }
    }
}

void print(int *array, int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%2d ", array[i]);
    }
    printf("\n");
}

void evaluateResult(int *array, int size)
{

    bool is_Sorted = true;
    for (int i = 0; i < size - 1; i++)
    {
        if (array[i] > array[i + 1])
        {
            is_Sorted = false;
            continue;
        }
    }

    if (is_Sorted)
    {
        printf("Sorted array\n");
    }
    else
        printf("Array is not sorted!!\n");
}