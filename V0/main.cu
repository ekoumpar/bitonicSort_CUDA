#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "bitonic.h"

int main() {
    int q = 10;
    size_t size = 1 << q ; // 2^q elements
    
    // Time variables
    hipEvent_t start, end;
    float exe_time;

    hipEventCreate(&start);
    hipEventCreate(&end);

    // Memory allocation
    int *GPU_array;                  
    hipError_t err = hipMallocManaged(&GPU_array,  size * sizeof(int));

    if (err != hipSuccess) {
      printf("Error by Memory allocation\n \"%s\"", hipGetErrorString(err));
      return -1;
    }
    
    srand(time(0));
    for (int i = 0; i < size; i++) {
      GPU_array[i] = rand() % 1000; // random numbers from (0-1000)
    }

    //print(GPU_array, size);
    hipEventRecord(start);

    // Bitonic sort in GPU
    bitonicSort(GPU_array, size);

    hipEventRecord(end);
    hipEventSynchronize(end);
    hipEventElapsedTime(&exe_time, start, end);

    //print(GPU_array, size);

    evaluateResult(GPU_array, size);
    printf("Execution time: %f ms", exe_time);
  
    hipFree(GPU_array);
    
    return 0;
}