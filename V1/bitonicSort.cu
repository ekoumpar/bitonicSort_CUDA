#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "bitonic.h"

__device__ void swap(int *array, int idx, int partner)
{
    int temp;
    temp = array[idx];
    array[idx] = array[partner];
    array[partner] = temp;
}

__global__ void initialExchange(int *array, int size)
{

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int group_size = 2; group_size <= 1024; group_size <<= 1)
    {
        for (int distance = group_size >> 1; distance > 0; distance >>= 1)
        {
            int idx = (tid / distance) * distance * 2 + (tid % distance);
            int partner = idx ^ distance;
            bool sort_descending = idx & group_size;

            if (idx < size && partner < size && idx < partner)
            { // ensure bounds are checked before accessing array

                if (!sort_descending && array[idx] > array[partner])
                {
                    // keep min elements
                    swap(array, idx, partner);
                }
                if (sort_descending && array[idx] < array[partner])
                {
                    // keep max elements
                    swap(array, idx, partner);
                }
            }

            __syncthreads();
        }
    }
}

__global__ void exchange_V0(int *array, int size, int group_size, int distance)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int idx = (tid / distance) * distance * 2 + (tid % distance);
    int partner = idx ^ distance;
    bool sort_descending = idx & group_size;

    if (idx < size && partner < size && idx < partner)
    { // ensure bounds are checked before accessing array

        if (!sort_descending && array[idx] > array[partner])
        {
            // keep min elements
            swap(array, idx, partner);
        }
        if (sort_descending && array[idx] < array[partner])
        {
            // keep max elements
            swap(array, idx, partner);
        }
    }
}

__global__ void exchange_V1(int *array, int size, int group_size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int distance = 1024; distance > 0; distance >>= 1)
    {
        int idx = (tid / distance) * distance * 2 + (tid % distance);
        int partner = idx ^ distance;
        bool sort_descending = idx & group_size;

        if (idx < size && partner < size && idx < partner)
        { // ensure bounds are checked before accessing array

            if (!sort_descending && array[idx] > array[partner])
            {
                // keep min elements
                swap(array, idx, partner);
            }
            if (sort_descending && array[idx] < array[partner])
            {
                // keep max elements
                swap(array, idx, partner);
            }
        }

        __syncthreads();
    }
}

__host__ void bitonicSort(int *array, int size)
{
    // GPU PARAMETERS
    int threads_per_block = 1024;
    int blocks_per_grid = size / threads_per_block;

    initialExchange<<<blocks_per_grid, threads_per_block>>>(array, size);

    for (int group_size = 2048; group_size <= size; group_size <<= 1)
    { // group_size doubles in each reccursion

        int distance = group_size >> 1;

        // Handle large distances (>1024)
        while (distance > 1024)
        {
            exchange_V0<<<blocks_per_grid, threads_per_block>>>(array, size, group_size, distance);

            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
                printf("CUDA Error: %s\n", hipGetErrorString(err));

            hipDeviceSynchronize();
            distance >>= 1;
        }

        // Handle small distances (<=1024)
        exchange_V1<<<blocks_per_grid, threads_per_block>>>(array, size, group_size);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
        }
    }
}

void print(int *array, int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%2d ", array[i]);
    }
    printf("\n");
}

void evaluateResult(int *array, int size)
{

    bool is_Sorted = true;
    for (int i = 0; i < size - 1; i++)
    {
        if (array[i] > array[i + 1])
        {
            is_Sorted = false;
            break;
        }
    }

    if (is_Sorted)
    {
        printf("Sorted array\n");
    }
    else
        printf("Array is not sorted!!\n");
}