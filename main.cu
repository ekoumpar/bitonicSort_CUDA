#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "bitonic.h"

int main(int argc, char *argv[])
{
  int q = atoi(argv[1]);
  size_t size = 1 << q; // 2^q elements

  // Time variables
  hipEvent_t start, end;
  float exe_time;

  hipEventCreate(&start);
  hipEventCreate(&end);

  // Memory allocation
  int *CPU_array = (int *)malloc(size * sizeof(int));
  int *GPU_array;

  hipError_t err = hipMalloc(&GPU_array, size * sizeof(int));
  if (err != hipSuccess)
  {
    printf("Error by Memory allocation\n \"%s\"", hipGetErrorString(err));
    return -1;
  }

  srand(time(0));
  for (int i = 0; i < size; i++)
  {
    CPU_array[i] = rand() % 1000; // random numbers from (0-1000)
  }

  // print(GPU_array, size);
  hipEventRecord(start);

  hipMemcpy(GPU_array, CPU_array, size * sizeof(int), hipMemcpyHostToDevice);

  // Bitonic sort in GPU
  bitonicSort(GPU_array, size);

  hipMemcpy(CPU_array, GPU_array, size * sizeof(int), hipMemcpyDeviceToHost);

  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&exe_time, start, end);

  // print(GPU_array, size);

  evaluateResult(CPU_array, size);
  printf("Execution time: %f s", exe_time / 1000);

  hipEventDestroy(start);
  hipEventDestroy(end);

  hipFree(GPU_array);
  free(CPU_array);

  return 0;
}